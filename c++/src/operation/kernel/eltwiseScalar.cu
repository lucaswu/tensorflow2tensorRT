#include "hip/hip_runtime.h"
#include "gpu_kernel.h"

NAME_SPACE_BEGIN
template<typename T1,typename T2>
__global__ void eltwiseScalarKernel_gpu(T1*pSrc,T2*pDst,int length,float scalar_value,int type )
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < length){
        T1 data = pSrc[idx];
        switch(type){
            case 0: pDst[idx] = data + scalar_value; break;
            case 1: pDst[idx] = data - scalar_value; break;
            case 2: pDst[idx] = data * scalar_value; break;
            case 3: pDst[idx] = data / scalar_value; break;
            case 4: pDst[idx] = fmin(data,scalar_value);break;
            case 5: pDst[idx] = fmax(data,scalar_value);break;
            case 7: pDst[idx] = fabs(data);break;
            case 8: 
                {   
                    T1 diff = data - scalar_value;
                    pDst[idx] = diff*diff;break;
                }    
            case 9: pDst[idx] = pow(data,scalar_value);break;
        }
         
    }
}

template<typename T1, typename T2>
result eltwiseScalar_gpu(T1* pSrc,T2* pDst,int length,float scalar_value,int type)
{
    auto ret = Ret_Success;

    CHECK_PTR(pSrc);
    CHECK_PTR(pDst);

    dim3 threadsPerBlock(512);
    dim3 blockSPerGrid(GWS(length, threadsPerBlock.x));

    switch(type){
        case 0:
        case 1:
        case 2:
        case 3:
        case 4:
        case 5:
        case 8:
        case 7:
        case 9:break;
        default:
            return Ret_NoSupportErr;
    };

    eltwiseScalarKernel_gpu<<<blockSPerGrid,threadsPerBlock>>>(pSrc,pDst,length,scalar_value,type);
    return ret;
}


template result eltwiseScalar_gpu(float* ,float* ,int ,float ,int );


NAME_SPACE_END
#include "hip/hip_runtime.h"
#include "gpu_kernel.h"
NAME_SPACE_BEGIN

template<typename T1,typename T2>
__global__ void castkernel_gpu(T1 *pSrc,T2*pDst,int length)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < length){
        pDst[idx] = pSrc[idx];
    }
}

template<typename T1, typename T2>
result cast_gpu(T1 *pSrc,T2*pDst,int length)
{
    result ret = Ret_Success;

    // CHECK_PTR(pSrc);
    // CHECK_PTR(pDst);

    dim3 threadsPerBlock(512);
    dim3 blockSPerGrid(GWS(length, threadsPerBlock.x));

    castkernel_gpu<<<blockSPerGrid,threadsPerBlock>>>(pSrc,pDst,length);
    return ret;
}

template result cast_gpu(unsigned char*,float* ,int);
template result cast_gpu(float*,unsigned char*, int);

NAME_SPACE_END